#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"

__global__ void CUDA_PixelShifting(PtrStep<unsigned char> src, PtrStep<unsigned char> depth, PtrStep<signed short> dst,
	int rows, int cols, int channels) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < cols && row < rows) {
		int src_offset = (row * src.step + channels * col);
		int dst_offset = (row * dst.step / sizeof(signed short) + channels * col);
		int depth_offset = (row * depth.step + col);

		int dis = (int)(depth[depth_offset] * 70. / 255.);
		//int dis = (int)(depth[depth_offset] * 70.);

		if (col > dis) {
			dst[dst_offset - (dis * channels) + 0] = src[src_offset + 0];
			dst[dst_offset - (dis * channels) + 1] = src[src_offset + 1];
			dst[dst_offset - (dis * channels) + 2] = src[src_offset + 2];
		}
	}
}

__global__ void CUDA_ImagePainting(PtrStep<signed short> img, int rows, int cols, int channels) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < cols && row < rows) {
		int rgb_offset = (row * img.step / sizeof(signed short) + channels * col);
		if (img[rgb_offset + 0] == -1 && img[rgb_offset + 1] == -1 && img[rgb_offset + 2] == -1) {
			for (int offset = 1; offset < 70; offset++) {

				if (col - offset >= 0 && img[rgb_offset - (offset * channels) + 0] != -1 && img[rgb_offset - (offset * channels) + 1] != -1 && img[rgb_offset - (offset * channels) + 2] != -1) {
					img[rgb_offset + 0] = img[rgb_offset - (offset * channels) + 0];
					img[rgb_offset + 1] = img[rgb_offset - (offset * channels) + 1];
					img[rgb_offset + 2] = img[rgb_offset - (offset * channels) + 2];
					break;
				}
				if (col + offset <= cols && img[rgb_offset + (offset * channels) + 0] != -1 && img[rgb_offset + (offset * channels) + 1] != -1 && img[rgb_offset + (offset * channels) + 2] != -1) {
					img[rgb_offset + 0] = img[rgb_offset + (offset * channels) + 0];
					img[rgb_offset + 1] = img[rgb_offset + (offset * channels) + 1];
					img[rgb_offset + 2] = img[rgb_offset + (offset * channels) + 2];
					break;
				}
			}
		}
	}
}

__global__ void CUDA_Concate(PtrStep<unsigned char> src1, PtrStep<unsigned char> src2, PtrStep<unsigned char> dst,
	int rows, int cols, int channels) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < cols && row < rows && col % 2 == 0) {
		int src_rgb_offset = (row * src1.step + channels * col);

		if (col <= cols) {
			int dst_rgb_offset = (row * src1.step + channels * col / 2);
			dst[dst_rgb_offset + 0] = src1[src_rgb_offset + 0];
			dst[dst_rgb_offset + 1] = src1[src_rgb_offset + 1];
			dst[dst_rgb_offset + 2] = src1[src_rgb_offset + 2];

			dst_rgb_offset += channels * (int)ceil(cols / 2.);
			dst[dst_rgb_offset + 0] = src2[src_rgb_offset + 0];
			dst[dst_rgb_offset + 1] = src2[src_rgb_offset + 1];
			dst[dst_rgb_offset + 2] = src2[src_rgb_offset + 2];
		}
	}
}

void PixelShifting(PtrStep<unsigned char> src, PtrStep<unsigned char> depth, PtrStep<signed short> dst,
	int height, int width, int channels) {
	const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
	const dim3 dimBlock(16, 16);
	CUDA_PixelShifting << <dimGrid, dimBlock >> > (src, depth, dst, height, width, channels);
}

void ImagePainting(PtrStep<signed short> img, int height, int width, int channels) {
	const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
	const dim3 dimBlock(16, 16);
	CUDA_ImagePainting << <dimGrid, dimBlock >> > (img, height, width, channels);
}

void ImageConcate(PtrStep<unsigned char> src1, PtrStep<unsigned char> src2, PtrStep<unsigned char> dst, int height, int width,
	int channels) {
	const dim3 dimGrid((int)ceil(width / 16.), (int)ceil(height / 16.));
	const dim3 dimBlock(16, 16);
	CUDA_Concate << <dimGrid, dimBlock >> > (src1, src2, dst, height, width, channels);
}
